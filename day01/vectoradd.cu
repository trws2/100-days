
#include <hip/hip_runtime.h>
#include<iostream>
#include<math.h>

// addition kernel
__global__
void add(int n, float *x, float *y) {
    int index = threadIdx.x; // the index of current thread within its block
    int stride = blockDim.x; // the no. of threads in the block
    for (int i = index; i < n; i += stride) {
        // printf("thread %d", index);
        // printf("threads in block %d", stride);
        y[i] = x[i] + y[i];
    }
}

int main(void) {
    int N = 1<<20; // 1 million
    float *x, *y;

    // allocate unified memory - accessible from GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for(int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }


    // run the kernel 
    add<<<1, 256>>>(N, x, y); // <<<numBlocks, blockSize>>>

    hipDeviceSynchronize();

    float maxError = 0.0f;
    for(int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i]-2.7f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // free memory
    hipFree(x);
    hipFree(y);

    return 0;
}